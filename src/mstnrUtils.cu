#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <R.h>

#include "mstnrUtils.h"


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

