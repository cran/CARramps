#include "hip/hip_runtime.h"
// kronVectMultForR.cu
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <R.h>
#include "mstnrUtils.h"
#include "kronVectMult1colForR_d.h"

#define BLOCKSIZE 16

void doKronVectMult1colD( double *a, double *b, double *c, double *retvect, int *na1, 
      int *nb1, int *nc1) 
{

   __global__ void kronVectMult1colOnDevice(double *a, double *b, double *c, 
   double *d, int na, int nb) ;

  void checkCUDAError(const char *msg) ;

  double *a_d, *b_d, *c_d, *d_d ;  // pointer to device memory
  int i, na = na1[0],nb = nb1[0], nab = na1[0] * nb1[0], nc = nc1[0];

  size_t sizea = na * na*sizeof(double);
  size_t sizeb = nb * nb*sizeof(double);
  size_t sizec = nab * sizeof(double); // Changed from mat ver


  // allocate array on device 
  hipMalloc((void **) &a_d, sizea);
  hipMalloc((void **) &b_d, sizeb);
  hipMalloc((void **) &c_d, sizec);
  hipMalloc((void **) &d_d, sizec);


  // copy data from host to device
  hipMemcpy(a_d, a, sizeof(double)*na*na, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, sizeof(double)*nb*nb, hipMemcpyHostToDevice);

  // Compute execution configuration
  // Changed from matrix version

  int threadx = min(na, BLOCKSIZE), thready = min(nb, BLOCKSIZE) ;
  int blockx = (na)/threadx + ((na)%threadx ==0?0:1) ;
  int blocky = nb/thready + (nb%thready ==0?0:1) ;

  // printf(" %d %d %d %d \n", threadx, thready, blockx, blocky ) ;

  dim3 threadsPerBlock( threadx, thready );   // block dim
  dim3 numBlocks(blockx, blocky);             // grid dim


for(i = 0; i < nc; i++)
{
  hipMemcpy(c_d, &c[i * nab], sizeof(double)*nab , hipMemcpyHostToDevice);
  // Check for any CUDA errors
    checkCUDAError("memcpy");
  hipMemset( d_d, 0, sizec ) ;
  // Check for any CUDA errors
    checkCUDAError("memset");


  // do calculation on device:

  // Call kronVectOnDevice kernel 

  kronVectMult1colOnDevice <<< numBlocks, threadsPerBlock >>> (a_d, b_d, c_d, d_d,
na, nb );

  // block until the device has completed
    hipDeviceSynchronize();

  // check if kernel execution generated an error
  // Check for any CUDA errors
    checkCUDAError("kernel invocation");

  // Retrieve result from device and store in c_hfd

  hipMemcpy(&retvect[i * nab], d_d, sizeof(double)*nab, hipMemcpyDeviceToHost);

  // Check for any CUDA errors
    checkCUDAError("memcpy");
}

  // clean up

  hipFree(a_d); hipFree(b_d); hipFree(c_d);  hipFree(d_d) ;
}

__global__ void kronVectMult1colOnDevice(double *a, double *b, double *c, 
double *d, int na, int nb)
{
  /*  a is na x na;  b is nb x nb;  c is (na*nb) x nc */

  double Csub = 0.0 ;  /* element computed by this thread */
  int N = na * nb,  acol,  bcol ;
  int arow = min(blockIdx.x*blockDim.x + threadIdx.x, na-1); /* output row */  
  int brow = min( blockIdx.y*blockDim.y + threadIdx.y, nb-1) ; /* output col */
  int idxtot = arow * nb + brow ;
  if( idxtot < N ) 
    {

      for( int k = 0; k < N; k++)
         {
           acol = k / nb ;
           bcol = k % nb ;
           Csub += a[ arow * na +  acol ] * b[brow * nb +  bcol ] 
               * c[k ] ;  
        }
      d[idxtot] = Csub ;
    }   
}

